#include "hip/hip_runtime.h"
#include <benchmark/cuda/kernels.hpp>


__global__
void kernel_ethash_v0()
{
}


__host__
bool ethash_v0(
        hipStream_t stream,
        uint32_t const blocks,
        uint32_t const threads)
{
    kernel_ethash_v0<<<blocks, threads, 0, stream>>>();
    CUDA_ER(hipStreamSynchronize(stream));
    CUDA_ER(hipGetLastError());

    return true;
}
